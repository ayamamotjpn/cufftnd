/* Copyright (c) 2020, Akiji Yamamoto. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining
 * a copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 * 
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 * 
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 * AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 * OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE SOFTWARE.
*/

#include <iostream>
#include <string>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>

#include "mtrns_gpu2.h"

extern "C" {

void testwt_c1(hipComplex *x, int n) {
	for (int i=0; i<5; i++) {
		std::cout<< x[i].x <<" "<< x[i].y <<" ";
	}
	printf("\n");
	for (int i=n-5; i<n; i++) {
		std::cout<< x[i].x <<" "<< x[i].y <<" ";
	}
	printf("\n");
}

void testwt_z1(hipDoubleComplex *x, int n) {
	for (int i=0; i<5; i++) {
		std::cout<< x[i].x <<" "<<x[i].x<<" ";
	}
	printf("\n");
	for (int i=n-5; i<n; i++) {
		std::cout<< x[i].x <<" "<< x[i].y<<" ";
	}
	printf("\n");
}

void testout(int dim, int dim0, int dim1, int mode, int nt, int *n, int *n0, int *ni) {
	std::cout << "dim " << dim << " mode " << mode 	<< " nt " << nt << "\n";
	std::cout << "n ";
	for (int i=0; i<dim; i++) {
		std::cout << n[i]<<" ";
	}
	std::cout << "\n";
	std::cout << "n0 ";
	for (int i=0; i<dim0; i++) {
		std::cout << n0[i]<<" ";
	}
	std::cout << "\n";

	std::cout << "ni ";
	for (int i=0; i<dim1; i++) {
		std::cout << ni[i]<<" ";
	}
	std::cout << "\n";
}

int get_nt(int *n, int dim) {
	int nt = 1;
	for (int i = 0; i < dim; i++) {
		nt *= n[i];
	}
	return nt;
}

void getn0i(int *n,int *n0, int *ni, int dim, int nt) {
	for (int i = 0; i < dim; i++) {
		n0[i] = n[i];
		ni[i] = nt / n0[i];
	}
}

// nd fft using cufft 1d and transpose matrix
void cufftnd_c(hipComplex* x, int dim, int n[], int mode) {
	// x should be a complex 1D array of size n[0]*n[1]..n[idm-1]
	// dim is the dimension
	// array indices of n should be a fortran order
	// mode 0 for forward 1 for inverse transformation
	int ni[dim];
	int n0[dim];

	int nt = get_nt(n,dim);
	getn0i(n, n0, ni, dim,nt);

	//std::cout << "x in cufftnd_c\n";
	//testwt_c1(x, nt);  // for test

	//testout(dim, dim, dim, mode, nt, n, n0, ni);

	hipComplex* x_d;
	//hipComplex *y_d;
	hipComplex *z_d;
	const hipComplex alpha={1.0,0.0};
	const hipComplex beta={0.0,0.0};
	hipblasHandle_t handle;
	hipblasCreate(&handle);

	int memsize = sizeof(hipComplex) * nt;
	hipMalloc((void **) &x_d, memsize);
	//hipMalloc((void **) &y_d, memsize);
	hipMalloc((void **) &z_d, memsize);  // for transpose
	hipMemcpy(x_d, x, memsize, hipMemcpyHostToDevice); // copy x to GPU mem
	hipfftHandle plan;

	for (int i = 0; i < dim; i++) {
		hipfftPlan1d(&plan, n[i], HIPFFT_C2C, ni[i]);        // 1D complex Fourier transformation
		if(mode==1) {
		    hipfftExecC2C(plan, x_d, x_d, HIPFFT_FORWARD);   // x_d : Fourier coefficients
		} else if(mode==-1) {
			hipfftExecC2C(plan, x_d, x_d, HIPFFT_BACKWARD);   // x_d : Fourier coefficients
		}
		hipDeviceSynchronize();
		hipMemcpy(x, x_d, memsize, hipMemcpyDeviceToHost); // copy back x_d to CPU mem
		//testwt_c1(x, nt);  // for test

		hipblasCgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, ni[i], n0[i], &alpha,
			x_d, n0[i], &beta, z_d, ni[i], z_d, ni[i]); // z_d : transpose of x_d
		//x_d = z_d;  // pointer copy
		hipMemcpy(x_d, z_d, memsize, hipMemcpyDeviceToDevice); // copy z_d to x_d
	}
	//std::cout <<"end of for loop\n";
	hipMemcpy(x, x_d, memsize, hipMemcpyDeviceToHost); // copy back x_d to CPU mem
	hipFree(x_d);
	//hipFree(y_d);
	hipFree(z_d);
}


// nd fft using cufft 1d and transpose matrix
void cufftnd_z(hipDoubleComplex* x, int dim, int n[], int mode) {
	// x should be a hipDoubleComplex 1D array of size n[0]*n[1]..n[idm-1]
	// dim is the dimension
	// array indices of n should be a fortran order
	// mode 0 for forward 1 for inverse transformation
	//hipfftResult istat;
	int ni[dim];
	int n0[dim];
	int nt = get_nt(n,dim);
	getn0i(n, n0, ni, dim, nt);

	//std::cout << "x in cufftnd_z\n";
	//testwt_z1(x, nt);  // for test

	//testout(dim, dim, dim, mode, nt, n, n0, ni);

	hipDoubleComplex* x_d;
	//hipDoubleComplex *y_d;
	hipDoubleComplex *z_d;
	const hipDoubleComplex alpha={1.0,0.0};
	const hipDoubleComplex beta={0.0,0.0};
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	int memsize = sizeof(hipDoubleComplex) * nt;
	hipMalloc((void **) &x_d, memsize);
	//hipMalloc((void **) &y_d, memsize);
	hipMalloc((void **) &z_d, memsize);  // for transpose
	hipMemcpy(x_d, x, memsize, hipMemcpyHostToDevice); // copy x to GPU mem
	hipfftHandle plan;
	for (int i = 0; i < dim; i++) {
		hipfftPlan1d(&plan,n[i], HIPFFT_Z2Z, ni[i]); // 1D hipDoubleComplex Fourier transformation
		if(mode==1) {
		    hipfftExecZ2Z(plan, x_d, x_d, HIPFFT_FORWARD);   // x_d : Fourier coefficients
		} else if(mode==-1) {
			hipfftExecZ2Z(plan, x_d, x_d, HIPFFT_BACKWARD);   // x_d : inverse Fourier coefficients
		}
		hipDeviceSynchronize();
		hipMemcpy(x, x_d, memsize, hipMemcpyDeviceToHost); // copy back x_d to CPU mem
		//testwt_z1(x, nt);  // for test

		//std::cout << i << " dim " << dim <<"\n";
		hipblasZgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, ni[i], n0[i], &alpha,
			x_d, n0[i], &beta, z_d, ni[i], z_d, ni[i]); // z_d : transpose of x_d
		//x_d = z_d;  // pointer copy
		hipMemcpy(x_d, z_d, memsize, hipMemcpyDeviceToDevice); // copy z_d to x_d
	}
	//std::cout <<"end of for loop\n";
	hipMemcpy(x, x_d, memsize, hipMemcpyDeviceToHost); // copy back x_d to CPU mem
	hipFree(x_d);
	//hipFree(y_d);
	hipFree(z_d);
}

void get_4d_ni0(int *n,int *n0,int *ni,int dim,int nt) {
	for (int i = 0; i < dim/2; i++) {
		n0[i] = n[i*2]*n[i*2+1];
		ni[i] = nt / n0[i];
	}
}

// 2^m d fft using cufft 2d and transpose matrix
void cufft2md_c(hipComplex* x, int dim, int n[], int mode) {
	// x should be a complex 1D array of size n[0]*n[1]..n[idm-1]
	// dim is the dimension
	// array indices of n should be a fortran order
	// mode 0 for forward 1 for inverse transformation

	int ni[dim];
	int n0[dim];

	//hipfftResult istat;
	int nt = get_nt(n,dim);
	get_4d_ni0(n,n0,ni,dim,nt);

	//testout(dim, 2, 2, mode, nt, n, n0, ni);

	hipComplex *x_d;
	hipComplex *y_d;
	hipComplex *z_d;
	const hipComplex alpha={1.0,0.0};
	const hipComplex beta={0.0,0.0};
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	int memsize = sizeof(hipComplex) * nt;
	hipMalloc((void **) &x_d, memsize);
	hipMalloc((void **) &y_d, memsize);
	hipMalloc((void **) &z_d, memsize);  // for transpose
	hipMemcpy(x_d, x, memsize, hipMemcpyHostToDevice); // copy x to GPU mem
	hipfftHandle plan;
	for (int i = 0; i < dim/2; i++) {
		hipfftPlanMany(&plan, 2, &n[i*2],NULL,1,0,NULL,1,0,HIPFFT_C2C, ni[i]);  // 2D complex Fourier transformation
		if(mode==1) {
		    hipfftExecC2C(plan, x_d, y_d, HIPFFT_FORWARD);   // y_d : Fourier coefficients of x_d
		} else if(mode==-1) {
			hipfftExecC2C(plan, x_d, y_d, HIPFFT_BACKWARD);   // y_d : inverse Fourier coefficients of x_d
		}
		hipDeviceSynchronize();
		hipMemcpy(x, y_d, memsize, hipMemcpyDeviceToHost); // copy back y_d to CPU mem
		//testwt_c1(x, nt);  // for test

		//std::cout<< "i "<< i << " dim/2 " << dim/2 <<"\n";
		hipblasCgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, ni[i], n0[i], &alpha,
			y_d, n0[i], &beta, z_d, ni[i], z_d, ni[i]); // z_d : transpose of y_d
		hipMemcpy(x_d, z_d, memsize, hipMemcpyDeviceToDevice); // copy z_d to x_d
	}
	hipMemcpy(x, x_d, memsize, hipMemcpyDeviceToHost); // copy back x_d to CPU mem
	hipFree(x_d);
	hipFree(y_d);
	hipFree(z_d);
}

// 2^m d fft using cufft 2d and transpose matrix
void cufft2md_z(hipDoubleComplex* x, int dim, int n[], int mode) {
	// x should be a hipDoubleComplex 1D array of size n[0]*n[1]..n[idm-1]
	// dim is the dimension
	// array indices of n should be a fortran order
	// mode 0 for forward 1 for inverse transformation
	int ni[dim];
	int n0[dim];

	int nt = get_nt(n,dim);
	get_4d_ni0(n,n0,ni,dim,nt);
	//testout(dim, 2, 2, mode, nt, n, n0, ni);

	hipDoubleComplex *x_d;
	hipDoubleComplex *y_d;
	hipDoubleComplex *z_d;
	const hipDoubleComplex alpha={1.0,0.0};
	const hipDoubleComplex beta={0.0,0.0};
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	int memsize = sizeof(hipDoubleComplex) * nt;
	hipMalloc((void **) &x_d, memsize);
	hipMalloc((void **) &y_d, memsize);
	hipMalloc((void **) &z_d, memsize);  // for transpose
	hipMemcpy(x_d, x, memsize, hipMemcpyHostToDevice); // copy x to GPU mem
	hipfftHandle plan;
	for (int i = 0; i < dim/2; i++) {
		hipfftPlanMany(&plan, 2, &n[i*2],NULL,1,0,NULL,1,0,HIPFFT_Z2Z,ni[i]); //2D hipDoubleComplex Fourier transformation
		if(mode==1) {
		    hipfftExecZ2Z(plan, x_d, y_d, HIPFFT_FORWARD);   // y_d : Fourier coefficients of x_d
		} else if(mode==-1) {
			hipfftExecZ2Z(plan, x_d, y_d, HIPFFT_BACKWARD);   // y_d : inverse Fourier coefficients of x_d
		}
		hipDeviceSynchronize();
		hipMemcpy(x, y_d, memsize, hipMemcpyDeviceToHost); // copy back x_d to CPU mem
		//testwt_z1(x, nt);  // for test

		hipblasZgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, ni[i], n0[i], &alpha,
			y_d, n0[i], &beta, z_d, ni[i], z_d, ni[i]); // z_d : transpose of y_d
		hipMemcpy(x_d, z_d, memsize, hipMemcpyDeviceToDevice); // copy z_d to x_d
	}
	hipMemcpy(x, x_d, memsize, hipMemcpyDeviceToHost); // copy back x_d to CPU mem
	hipFree(x_d);
	hipFree(y_d);
	hipFree(z_d);
}


void get_5d_n0i(int *n,int *n0,int *ni,int dim,int nt) {
	n0[0] = n[0]*n[1]*n[2];
	n0[1] = n[3]*n[4];
	ni[0] = nt / n0[0];
	ni[1] = nt / n0[1];
}

void get_idim(int *idim) {
	idim[0]=3;
	idim[1]=2;
}

/*
	n0[0] = n[0]*n[1]*n[2];
	n0[1] = n[3]*n[4];
	ni[0] = nt / n0[0];
	ni[1] = nt / n0[1];
*/

// 5d  fft using cufft 3d, 2d  and transpose matrix
void cufft5d_c(hipComplex* x, int n[], int mode) {
	// x should be a complex 1D array of size n[0]*n[1]..n[idm-1]
	// array indices of n should be a fortran order
	// mode 0 for forward 1 for inverse transformation
	int dim=5;

	int ni[2];
	int n0[2];
	int idim[2];

	get_idim(idim);
	int nt = get_nt(n,dim);
	get_5d_n0i(n,n0,ni,dim,nt);
	//testout(dim, 2, 2, mode, nt, n, n0, ni);

	hipComplex *x_d;
	hipComplex *y_d;
	hipComplex *z_d;
	const hipComplex alpha={1.0,0.0};
	const hipComplex beta={0.0,0.0};
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	int memsize = sizeof(hipComplex) * nt;
	hipMalloc((void **) &x_d, memsize);
	hipMalloc((void **) &y_d, memsize);
	hipMalloc((void **) &z_d, memsize);  // for transpose
	hipMemcpy(x_d, x, memsize, hipMemcpyHostToDevice); // copy x to GPU mem
	hipfftHandle plan;
	for (int i = 0; i < 2; i++) {
		hipfftPlanMany(&plan, idim[i], &n[i*3],NULL,1,0,NULL,1,0,HIPFFT_Z2Z,ni[i]); //2D  complex Fourier transformation
		if(mode==1) {
		    hipfftExecC2C(plan, x_d, y_d, HIPFFT_FORWARD);   // y_d : Fourier coefficients of x_d
		} else if(mode==-1) {
			hipfftExecC2C(plan, x_d, y_d, HIPFFT_BACKWARD);   // y_d : inverse Fourier coefficients of x_d
		}
		hipDeviceSynchronize();
		hipblasCgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, ni[i], n0[i], &alpha,
			y_d, n0[i], &beta, z_d, ni[i], z_d, ni[i]); // z_d : transpose of y_d
		hipMemcpy(x_d, z_d, memsize, hipMemcpyDeviceToDevice); // copy z_d to x_d
	}
	hipMemcpy(x, x_d, memsize, hipMemcpyDeviceToHost); // copy back x_d to CPU mem
	hipFree(x_d);
	hipFree(y_d);
	hipFree(z_d);
}

// 5d  fft using cufft 3d, 2d  and transpose matrix
void cufft5d_z(hipDoubleComplex* x, int n[], int mode) {
	// x should be a hipDoubleComplex 1D array of size n[0]*n[1]..n[idm-1]
	// array indices of n should be a fortran order
	// mode 0 for forward 1 for inverse transformation
	int dim=5;
	int ni[2];
	int n0[2];
	int idim[2];

	get_idim(idim);
	int nt = get_nt(n,dim);
	get_5d_n0i(n,n0,ni,dim,nt);
	//testout(dim, 2, 2, mode, nt, n, n0, ni);

	hipDoubleComplex *x_d;
	hipDoubleComplex *y_d;
	hipDoubleComplex *z_d;
	const hipDoubleComplex alpha={1.0,0.0};
	const hipDoubleComplex beta={0.0,0.0};
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	int memsize = sizeof(hipDoubleComplex) * nt;
	hipMalloc((void **) &x_d, memsize);
	hipMalloc((void **) &y_d, memsize);
	hipMalloc((void **) &z_d, memsize);  // for transpose
	hipMemcpy(x_d, x, memsize, hipMemcpyHostToDevice); // copy x to GPU mem
	hipfftHandle plan;
	for (int i = 0; i < 2; i++) {
		hipfftPlanMany(&plan, idim[i], &n[i*3],NULL,1,0,NULL,1,0,HIPFFT_Z2Z,ni[i]); //2D hipDoubleComplex Fourier transformation
		if(mode==1) {
		    hipfftExecZ2Z(plan, x_d, y_d, HIPFFT_FORWARD);   // y_d : Fourier coefficients of x_d
		} else if(mode==-1) {
			hipfftExecZ2Z(plan, x_d, y_d, HIPFFT_BACKWARD);   // y_d : inverse Fourier coefficients of x_d
		}
		hipDeviceSynchronize();
		hipblasZgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, ni[i], n0[i], &alpha,
			y_d, n0[i], &beta, z_d, ni[i], z_d, ni[i]); // z_d : transpose of y_d
		hipMemcpy(x_d, z_d, memsize, hipMemcpyDeviceToDevice); // copy z_d to x_d
	}
	hipMemcpy(x, z_d, memsize, hipMemcpyDeviceToHost); // copy back x_d to CPU mem
	hipFree(x_d);
	hipFree(y_d);
	hipFree(z_d);
}


void get_3md_n0i(int *n,int *n0,int *ni,int dim,int nt) {
	for (int i = 0; i < dim/3; i++) {
		n0[i] = n[i*3]*n[i*3+1]*n[i*3+2];
		ni[i] = nt / n0[i];
	}
}

// 3^m d fft using cufft 3d and transpose matrix
void cufft3md_c(hipComplex* x, int dim, int n[], int mode) {
	// x should be a complex 1D array of size n[0]*n[1]..n[idm-1]
	// dim is the dimension
	// array indices of n should be a fortran order
	// mode 0 for forward 1 for inverse transformation
	int ni[dim];
	int n0[dim];

	//hipfftResult istat;
	int nt = get_nt(n,dim);
	get_3md_n0i(n,n0,ni,dim,nt);
	//testout(dim, 2, 2, mode, nt, n, n0, ni);

	hipComplex *x_d;
	hipComplex *y_d;
	hipComplex *z_d;
	const hipComplex alpha={1.0,0.0};
	const hipComplex beta={0.0,0.0};
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	int memsize = sizeof(hipComplex) * nt;
	hipMalloc((void **) &x_d, memsize);
	hipMalloc((void **) &y_d, memsize);
	hipMalloc((void **) &z_d, memsize);  // for transpose
	hipMemcpy(x_d, x, memsize, hipMemcpyHostToDevice); // copy x to GPU mem
	hipfftHandle plan;
	for (int i = 0; i < dim/3; i++) {
		hipfftPlanMany(&plan, 3, &n[i*3],NULL,1,0,NULL,1,0,HIPFFT_C2C, ni[i]);  // 3D complex Fourier transformation
		if(mode==1) {
		    hipfftExecC2C(plan, x_d, y_d, HIPFFT_FORWARD);   // y_d : Fourier coefficients of x_d
		} else if(mode==-1) {
			hipfftExecC2C(plan, x_d, y_d, HIPFFT_BACKWARD);   // y_d : inverse Fourier coefficients of x_d
		}
		hipDeviceSynchronize();
		hipblasCgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, ni[i], n0[i], &alpha,
			y_d, n0[i], &beta, z_d, ni[i], z_d, ni[i]); // z_d : transpose of y_d
		hipMemcpy(x_d, z_d, memsize, hipMemcpyDeviceToDevice); // copy z_d to x_d
	}
	hipMemcpy(x, x_d, memsize, hipMemcpyDeviceToHost); // copy back x_d to CPU mem
	hipFree(x_d);
	hipFree(y_d);
	hipFree(z_d);
}

// 3^m d fft using cufft 3d and transpose matrix
void cufft3md_z(hipDoubleComplex* x, int dim, int n[], int mode) {
	// x should be a hipDoubleComplex 1D array of size n[0]*n[1]..n[idm-1]
	// dim is the dimension
	// array indices of n should be a fortran order
	// mode 0 for forward 1 for inverse transformation
	int ni[dim];
	int n0[dim];
	int nt = get_nt(n,dim);
	get_3md_n0i(n,n0,ni,dim,nt);
	//testout(dim, 2, 2, mode, nt, n, n0, ni);

	hipDoubleComplex *x_d;
	hipDoubleComplex *y_d;
	hipDoubleComplex *z_d;
	const hipDoubleComplex alpha={1.0,0.0};
	const hipDoubleComplex beta={0.0,0.0};
	hipblasHandle_t handle;
	hipblasCreate(&handle);
	int memsize = sizeof(hipDoubleComplex) * nt;
	hipMalloc((void **) &x_d, memsize);
	hipMalloc((void **) &y_d, memsize);
	hipMalloc((void **) &z_d, memsize);  // for transpose
	hipMemcpy(x_d, x, memsize, hipMemcpyHostToDevice); // copy x to GPU mem
	hipfftHandle plan;
	for (int i = 0; i < dim/3; i++) {
		hipfftPlanMany(&plan, 3, &n[i*3],NULL,1,0,NULL,1,0,HIPFFT_Z2Z,ni[i]); //3D hipDoubleComplex Fourier transformation
		if(mode==1) {
		    hipfftExecZ2Z(plan, x_d, y_d, HIPFFT_FORWARD);   // y_d : Fourier coefficients of x_d
		} else if (mode==-1) {
			hipfftExecZ2Z(plan, x_d, y_d, HIPFFT_BACKWARD);   // y_d : inverse Fourier coefficients of x_d
		}
		hipDeviceSynchronize();
		hipblasZgeam(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, ni[i], n0[i], &alpha,
			y_d, n0[i], &beta, z_d, ni[i], z_d, ni[i]); // z_d : transpose of y_d
		hipMemcpy(x_d, z_d, memsize, hipMemcpyDeviceToDevice); // copy z_d to x_d
	}
	hipMemcpy(x, x_d, memsize, hipMemcpyDeviceToHost); // copy back x_d to CPU mem
	hipFree(x_d);
	hipFree(y_d);
	hipFree(z_d);
}

// wrapper for 4D fft
void cufft4d_c(hipComplex* x, int n[], int mode) {
	cufft2md_c(x, 4, n, mode);
}

// wrapper for 4D fft
void cufft4d_z(hipDoubleComplex* x, int n[], int mode) {
	cufft2md_z(x, 4, n, mode);
}

// wrapper for 6D fft
void cufft6d_c(hipComplex* x, int n[], int mode) {
	cufft3md_c(x, 6, n, mode);
}

// wrapper for 6D fft
void cufft6d_z(hipDoubleComplex* x, int n[], int mode) {
	cufft3md_z(x, 6, n, mode);
}

}  // end of extern "C"